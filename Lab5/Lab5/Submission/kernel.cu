#include "hip/hip_runtime.h"
// Sandip Saha Joy

#include "hip/hip_runtime.h"
#include ""
#include <wb.h>

#define NUM_BINS 4096

#define CUDA_CHECK(ans)                                                   \
    { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line,
	bool abort = true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
			file, line);
		if (abort)
			exit(code);
	}
}


__global__ void histogram_kernel(unsigned int *input, unsigned int *bin, int inputLength, int binLength)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int x = i;
	while (x < inputLength)
	{
		atomicAdd(&(bin[input[x]]), 1);
		x += (blockDim.x * gridDim.x);
	}
	__syncthreads();
}


__global__ void histogram_kernel_optimized(unsigned int *input, unsigned int *bin, int inputLength, int binLength)
{
	int x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ unsigned int bin_shared[];

	x = threadIdx.x;
	while (x < binLength)
	{
		bin_shared[x] = 0;
		x += blockDim.x;
	}
	__syncthreads();

	x = i;
	while (x < inputLength)
	{
		atomicAdd(&(bin_shared[input[x]]), 1);
		x += (blockDim.x * gridDim.x);
	}
	__syncthreads();

	x = threadIdx.x;
	while (x < binLength)
	{
		atomicAdd(&(bin[x]), bin_shared[x]);
		x += blockDim.x;
	}
	__syncthreads();
}

__global__ void post_processing(unsigned int *bin, int binLength)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < binLength)
	{
		if (bin[i] > 127)
			bin[i] = 127;
	}
}

int main(int argc, char *argv[]) {
	wbArg_t args;
	int inputLength;
	int binLength = NUM_BINS;
	unsigned int *hostInput;
	unsigned int *hostBins;
	unsigned int *deviceInput;
	unsigned int *deviceBins;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput = (unsigned int *)wbImport(wbArg_getInputFile(args, 0),
		&inputLength, "Integer");
	hostBins = (unsigned int *)malloc(binLength * sizeof(unsigned int));
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The input length is ", inputLength);
	wbLog(TRACE, "The number of bins is ", binLength);

	wbTime_start(GPU, "Allocating GPU memory.");
	// TODO: Allocate GPU memory here
	hipMalloc(&deviceInput, inputLength * sizeof(unsigned int));
	hipMalloc(&deviceBins, binLength * sizeof(unsigned int));

	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	// TODO: Copy memory to the GPU here
	hipMemcpy(deviceInput, hostInput, inputLength * sizeof(unsigned int), hipMemcpyHostToDevice);

	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	// Launch kernel
	// ----------------------------------------------------------
	wbLog(TRACE, "Launching kernel");
	wbTime_start(Compute, "Performing CUDA computation");

	// TODO: Perform kernel computation here
	dim3 gridDim(32);
	dim3 blockDim(1024);

	//histogram_kernel << <gridDim, blockDim, binLength * sizeof(unsigned int) >> >(deviceInput, deviceBins, inputLength, binLength);
	histogram_kernel_optimized << <gridDim, blockDim, binLength * sizeof(unsigned int) >> >(deviceInput, deviceBins, inputLength, binLength);
	post_processing << <gridDim, blockDim, binLength * sizeof(unsigned int) >> >(deviceBins, binLength);

	// You should call the following lines after you call the kernel.
	CUDA_CHECK(hipGetLastError());
	CUDA_CHECK(hipDeviceSynchronize());

	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	// TODO: Copy the GPU memory back to the CPU here
	hipMemcpy(hostBins, deviceBins, binLength * sizeof(unsigned int), hipMemcpyDeviceToHost);

	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	// TODO: Free the GPU memory here
	hipFree(deviceInput);
	hipFree(deviceBins);

	wbTime_stop(GPU, "Freeing GPU Memory");

	// Verify correctness
	// -----------------------------------------------------
	wbSolution(args, hostBins, binLength);

	free(hostBins);
	free(hostInput);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}























































































































































































































































//Reference: https://bit.ly/2Hf6HKz