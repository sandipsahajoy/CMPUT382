#include "hip/hip_runtime.h"
// Sandip Saha Joy

#include "hip/hip_runtime.h"
#include ""

#include <wb.h>

#define TILE_WIDTH 16

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) 
{
	// TODO: Insert code to implement matrix multiplication 
	// here you have to use shared memory for this lab.
	// Take a the tiled matrix multiplication. Also we 
	// will be testing the speed up between a basic
	// matrix multiplication and this kernel. To pass 
	// the tests for the tiled matrix multiplication
	// you will need to have the correct output and
	// have a significant speed up over a basic matrix
	// multiplication.
	//
	// HINT: Take a look at the slides
	// HINT: Look at TILE_WIDTH defined above
	__shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
	__shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x, by = blockIdx.y;
	int tx = threadIdx.x, ty = threadIdx.y;
	int row = by * blockDim.y +ty;
	int	col = bx * blockDim.x + tx;
	float Cvalue = 0;

	for (int p = 0; p < (numAColumns - 1) / TILE_WIDTH + 1; p++) {
		if (row < numARows && p * TILE_WIDTH + tx < numAColumns)
			ds_M[ty][tx] = A[row * numAColumns + p * TILE_WIDTH + tx];
		else
			ds_M[ty][tx] = 0;
		if (col < numBColumns && p * TILE_WIDTH + ty < numBRows)
			ds_N[ty][tx] = B[(p * TILE_WIDTH + ty) * numBColumns + col];
		else
			ds_N[ty][tx] = 0;
		__syncthreads();
		for (int k = 0; k < TILE_WIDTH; ++k)
			Cvalue += ds_M[ty][k] * ds_N[k][tx];
		__syncthreads();
	}
	if (row < numCRows && col < numCColumns)
		C[row *numCColumns + col] = Cvalue;
	
}

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
	    }                                                                 \
    } while (0)

int main(int argc, char **argv) {
	wbArg_t args;
	float *hostA; // The A matrix
	float *hostB; // The B matrix
	float *hostC; // The output C matrix
	float *deviceA;
	float *deviceB;
	float *deviceC;
	int numARows;    // number of rows in the matrix A
	int numAColumns; // number of columns in the matrix A
	int numBRows;    // number of rows in the matrix B
	int numBColumns; // number of columns in the matrix B
	int numCRows;
	int numCColumns;

	args = wbArg_read(argc, argv);

#if LAB_DEBUG
	std::cout << "Running Tiled Matrix Multiplicaion ..." << std::endl;
#endif

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
		&numAColumns);
	hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
		&numBColumns);
	// TODO: Allocate the hostC matrix
	hostC = (float *)malloc(numARows * numBColumns * sizeof(float));
	
	wbTime_stop(Generic, "Importing data and creating memory on host");

	// TODO: Set numCRows and numCColumns
	numCRows = numARows;
	numCColumns = numBColumns;

	int sizeA = numARows * numAColumns * sizeof(float);
	int sizeB = numBRows * numBColumns * sizeof(float);
	int sizeC = numCRows * numCColumns * sizeof(float);

	wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
	wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
	wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);

	wbTime_start(GPU, "Allocating GPU memory.");
	// TODO: Allocate GPU memory here
	hipMalloc(&deviceA, sizeA);
	hipMalloc(&deviceB, sizeB);
	hipMalloc(&deviceC, sizeC);

	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	// TODO: Copy memory to the GPU here
	hipMemcpy(deviceA, hostA, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(deviceB, hostB, sizeB, hipMemcpyHostToDevice);

	wbTime_stop(GPU, "Copying input memory to the GPU.");

	// TODO: Initialize the grid and block dimensions here
	// Here you will have to use dim3
	//
	// HINT: Take a look at the slides
	// HINT: Look at TILE_WIDTH defined at the top
	//
	// dim3 blockDim( ... );
	// dim3 gridDim( ... );
	dim3 blockDim(32, 32);
	dim3 gridDim(32, 32);



	// wbLog(TRACE, "The block dimensions are ", blockDim.x, " x ", blockDim.y);
	// wbLog(TRACE, "The grid dimensions are ", gridDim.x, " x ", gridDim.y);

	wbTime_start(Compute, "Performing CUDA computation");
	// TODO:: Launch the GPU Kernel here
	matrixMultiplyShared<<<gridDim, blockDim>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns);

	hipDeviceSynchronize();
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	// TODO:: Copy the GPU memory back to the CPU here
	hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost);

	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	// TODO:: Free the GPU memory here
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);
	
	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, hostC, numCRows, numCColumns);

	free(hostA);
	free(hostB);
	free(hostC);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}
