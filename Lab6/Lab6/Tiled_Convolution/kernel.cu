#include "hip/hip_runtime.h"
// Sandip Saha Joy 
#include "hip/hip_runtime.h"
#include ""

#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
	    }                                                                     \
    } while (0)

#define Mask_width 5
#define Mask_radius Mask_width / 2
#define TILE_WIDTH 16
#define w (TILE_WIDTH + Mask_width - 1)
#define clamp(x) (min(max((x), 0.0), 1.0))

__global__ void tiled_convolution(float *I, const float *M, float *P, int channels, int width, int height) {
	//TODO: INSERT CODE HERE

	__shared__ float N_ds[w][w];
	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

	for (int channel = 0; channel < channels; channel++) 
	{
		int	ty = (threadIdx.y * TILE_WIDTH + threadIdx.x) / w;
		int tx = (threadIdx.y * TILE_WIDTH + threadIdx.x) % w;
		int row_o = blockIdx.y *TILE_WIDTH + ty;
		int col_o = blockIdx.x *TILE_WIDTH + tx;
		int row_i = row_o - Mask_radius;
		int col_i = col_o - Mask_radius;
		if (row_i >= 0 && row_i < height && col_i >= 0 && col_i < width)
			N_ds[ty][tx] = I[(row_i * width + col_i) * channels + channel];
		else
			N_ds[ty][tx] = 0;


		ty = (threadIdx.y * TILE_WIDTH + threadIdx.x + TILE_WIDTH * TILE_WIDTH) / w;
		tx = (threadIdx.y * TILE_WIDTH + threadIdx.x + TILE_WIDTH * TILE_WIDTH) % w;
		row_o = blockIdx.y *TILE_WIDTH + ty;
		col_o = blockIdx.x *TILE_WIDTH + tx;
		row_i = row_o - Mask_radius;
		col_i = col_o - Mask_radius;
		if (ty < w) 
		{
			if (row_i >= 0 && row_i < height && col_i >= 0 && col_i < width)
				N_ds[ty][tx] = I[(row_i * width + col_i) * channels + channel];
			else
				N_ds[ty][tx] = 0;
		}
		__syncthreads();


		float pixVal = 0;
		for (int row = 0; row < Mask_width; row++)
		{
			for (int col = 0; col < Mask_width; col++)
			{
				pixVal += N_ds[threadIdx.y + row][threadIdx.x + col] * M[row * Mask_width + col];
			}
		}

		if (Col < width  && Row < height)
		{
			P[(Row*width + Col)*channels + channel] = clamp(pixVal);
		}
		__syncthreads();
	}

}

int main(int argc, char *argv[]) {
	wbArg_t arg;
	int maskRows;
	int maskColumns;
	int imageChannels;
	int imageWidth;
	int imageHeight;
	char *inputImageFile;
	char *inputMaskFile;
	wbImage_t inputImage;
	wbImage_t outputImage;
	float *hostInputImageData;
	float *hostOutputImageData;
	float *hostMaskData;
	float *deviceInputImageData;
	float *deviceOutputImageData;
	float *deviceMaskData;

	arg = wbArg_read(argc, argv); /* parse the input arguments */

	inputImageFile = wbArg_getInputFile(arg, 0);
	inputMaskFile = wbArg_getInputFile(arg, 1);

	inputImage = wbImport(inputImageFile);
	hostMaskData = (float *)wbImport(inputMaskFile, &maskRows, &maskColumns);

	assert(maskRows == 5);    /* mask height is fixed to 5 in this mp */
	assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

	imageWidth = wbImage_getWidth(inputImage);
	imageHeight = wbImage_getHeight(inputImage);
	imageChannels = wbImage_getChannels(inputImage);

	outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

	hostInputImageData = wbImage_getData(inputImage);
	hostOutputImageData = wbImage_getData(outputImage);

	wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

	wbTime_start(GPU, "Doing GPU memory allocation");
	//TODO: INSERT CODE HERE
	hipMalloc(&deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
	hipMalloc(&deviceMaskData, maskRows * maskColumns * sizeof(float));
	hipMalloc(&deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));

	wbTime_stop(GPU, "Doing GPU memory allocation");

	wbTime_start(Copy, "Copying data to the GPU");
	//TODO: INSERT CODE HERE
	hipMemcpy(deviceInputImageData, hostInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(deviceMaskData, hostMaskData, maskRows * maskColumns * sizeof(float), hipMemcpyHostToDevice);

	wbTime_stop(Copy, "Copying data to the GPU");

	wbTime_start(Compute, "Doing the computation on the GPU");
	//TODO: INSERT CODE HERE
	dim3 dimGrid(ceil((float)imageWidth / TILE_WIDTH), ceil((float)imageHeight / TILE_WIDTH));
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

	tiled_convolution << <dimGrid, dimBlock >> >(deviceInputImageData, deviceMaskData, deviceOutputImageData, imageChannels, imageWidth, imageHeight);
	
	hipDeviceSynchronize();
	wbTime_stop(Compute, "Doing the computation on the GPU");

	wbTime_start(Copy, "Copying data from the GPU");
	//TODO: INSERT CODE HERE
	hipMemcpy(hostOutputImageData, deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyDeviceToHost);

	wbTime_stop(Copy, "Copying data from the GPU");

	wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

	wbSolution(arg, outputImage);

	//TODO: RELEASE CUDA MEMORY
	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);
	hipFree(deviceMaskData);

	free(hostMaskData);
	wbImage_delete(outputImage);
	wbImage_delete(inputImage);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}
