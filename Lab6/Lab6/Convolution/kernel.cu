#include "hip/hip_runtime.h"
// Sandip Saha Joy
#include "hip/hip_runtime.h"
#include ""

#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
		    }                                                                     \
      } while (0)

#define TILE_WIDTH 16
#define Mask_width 5
#define Mask_radius Mask_width / 2
#define clamp(x) (min(max((x), 0.0), 1.0))

__global__ void convolution(float *I, const float *M, float *P, int channels, int width, int height) {
	//TODO: INSERT CODE HERE

	int Col = blockIdx.x * blockDim.x + threadIdx.x;
	int Row = blockIdx.y * blockDim.y + threadIdx.y;

	if (Col < width  && Row < height)
	{
		for (int channel = 0; channel < channels; channel++)
		{
			float pixVal = 0;

			int N_start_row = Row - Mask_radius;
			int N_start_col = Col - Mask_radius;

			for (int j = 0; j < Mask_width; ++j)
			{
				for (int k = 0; k < Mask_width; ++k)
				{
					int curRow = N_start_row + j;
					int curCol = N_start_col + k;
					if (curRow >= -1 && curRow < height && curCol >= -1 && curCol < width)
					{
						pixVal += I[(curRow*width + curCol)*channels + channel] * M[j*Mask_width + k];
					}
				}
			}
			P[(Row*width + Col)*channels + channel] = clamp(pixVal);
		}
	}

}

int main(int argc, char *argv[]) {
	wbArg_t arg;
	int maskRows;
	int maskColumns;
	int imageChannels;
	int imageWidth;
	int imageHeight;
	char *inputImageFile;
	char *inputMaskFile;
	wbImage_t inputImage;
	wbImage_t outputImage;
	float *hostInputImageData;
	float *hostOutputImageData;
	float *hostMaskData;
	float *deviceInputImageData;
	float *deviceOutputImageData;
	float *deviceMaskData;

	arg = wbArg_read(argc, argv); /* parse the input arguments */

	inputImageFile = wbArg_getInputFile(arg, 0);
	inputMaskFile = wbArg_getInputFile(arg, 1);

	inputImage = wbImport(inputImageFile);
	hostMaskData = (float *)wbImport(inputMaskFile, &maskRows, &maskColumns);

	assert(maskRows == 5);    /* mask height is fixed to 5 in this mp */
	assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

	imageWidth = wbImage_getWidth(inputImage);
	imageHeight = wbImage_getHeight(inputImage);
	imageChannels = wbImage_getChannels(inputImage);

	outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

	hostInputImageData = wbImage_getData(inputImage);
	hostOutputImageData = wbImage_getData(outputImage);

	wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

	wbTime_start(GPU, "Doing GPU memory allocation");
	//TODO: INSERT CODE HERE
	hipMalloc(&deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
	hipMalloc(&deviceMaskData, maskRows * maskColumns * sizeof(float));
	hipMalloc(&deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));


	wbTime_stop(GPU, "Doing GPU memory allocation");

	wbTime_start(Copy, "Copying data to the GPU");
	//TODO: INSERT CODE HERE
	hipMemcpy(deviceInputImageData, hostInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(deviceMaskData, hostMaskData, maskRows * maskColumns * sizeof(float), hipMemcpyHostToDevice);

	wbTime_stop(Copy, "Copying data to the GPU");

	wbTime_start(Compute, "Doing the computation on the GPU");
	//TODO: INSERT CODE HERE
	dim3 dimGrid(ceil((float)imageWidth / TILE_WIDTH), ceil((float)imageHeight / TILE_WIDTH));
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

	convolution << <dimGrid, dimBlock >> >(deviceInputImageData, deviceMaskData, deviceOutputImageData, imageChannels, imageWidth, imageHeight);
	
	hipDeviceSynchronize();
	wbTime_stop(Compute, "Doing the computation on the GPU");

	wbTime_start(Copy, "Copying data from the GPU");
	//TODO: INSERT CODE HERE
	hipMemcpy(hostOutputImageData, deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyDeviceToHost);
	
	wbTime_stop(Copy, "Copying data from the GPU");

	wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

	wbSolution(arg, outputImage);

	//TODO: RELEASE CUDA MEMORY
	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);
	hipFree(deviceMaskData);
	
	free(hostMaskData);
	wbImage_delete(outputImage);
	wbImage_delete(inputImage);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}
