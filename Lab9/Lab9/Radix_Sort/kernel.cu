#include "hip/hip_runtime.h"
// Sandip Saha Joy
#include "hip/hip_runtime.h"
#include ""

#include <wb.h>

#define BLOCK_SIZE 512 //TODO: You can change this

#define wbCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void blockadd(int* g_aux, int* g_odata, int length){
	int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (blockIdx.x > 0 && index < length)
		g_odata[index] += g_aux[blockIdx.x];

}
__global__ void scan(int *g_odata, int *g_idata, int *g_aux, int length){

	int index = blockIdx.x*blockDim.x + threadIdx.x;
	__shared__ float temp[BLOCK_SIZE];

	if (index < length){
		temp[threadIdx.x] = g_idata[index];
	}

	for (int stride = 1; stride <= threadIdx.x; stride *= 2){
		__syncthreads();
		float k = 0.0;
		if (threadIdx.x >= stride){
			k = temp[threadIdx.x - stride];
		}
		__syncthreads();
		temp[threadIdx.x] += k;
	}

	__syncthreads();

	if (index + 1 < length){
		g_odata[index + 1] = temp[threadIdx.x];
	}
	g_odata[0] = 0;


	if (g_aux != NULL && threadIdx.x == blockDim.x - 1){
		g_aux[blockIdx.x] = g_odata[index + 1];
		g_odata[index + 1] = 0;
	}
}

void recursive_scan(int* deviceOutput, int* deviceInput, int numElements){
	int numBlocks = (numElements / BLOCK_SIZE) + 1;
	if (numBlocks == 1){
		dim3 block(BLOCK_SIZE, 1);
		dim3 grid(numBlocks, 1);
		scan << <grid, block >> >(deviceOutput, deviceInput, NULL, numElements);
		hipDeviceSynchronize();
	}
	else{
		int* deviceAux;
		hipMalloc((void**)&deviceAux, (numBlocks*sizeof(int)));
		dim3 block(BLOCK_SIZE, 1);
		dim3 grid(numBlocks, 1);
		scan << <grid, block >> >(deviceOutput, deviceInput, deviceAux, numElements);
		hipDeviceSynchronize();

		int *deviceAuxPass;
		hipMalloc((void**)&deviceAuxPass, (numBlocks*sizeof(int)));
		dim3 grid2(1, 1);
		dim3 block2(numBlocks, 1, 1);
		scan << <grid2, block2 >> >(deviceAuxPass, deviceAux, NULL, numBlocks);
		hipDeviceSynchronize();

		recursive_scan(deviceAuxPass, deviceAux, numBlocks);
		blockadd << <block2, block >> >(deviceAuxPass, deviceOutput, numElements);
		hipDeviceSynchronize();

		hipFree(deviceAux);
		hipFree(deviceAuxPass);
	}

}

__global__ void scatter(int *in_d, int *index_d, int *out_d, int length) {
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if (index < length)
		out_d[index_d[index]] = in_d[index];
	__syncthreads();
}

__global__ void split_A(int *in_d, int *out_d, int length, int bit_d) {
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	int bit = 0;
	if (index < length) {
		bit = in_d[index] & (1 << bit_d);
		if (bit > 0)
			bit = 1;
		else
			bit = 0;
		out_d[index] = 1 - bit;
	}
	__syncthreads();
}

__global__ void split_B(int *in_d, int *out_d, int length) {
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	int x = in_d[length - 1] + out_d[length - 1];
	__syncthreads();
	if (index < length && out_d[index] == 0) {
		__syncthreads();
		in_d[index] = index - in_d[index] + x;
	}
}

void sort(int* d_deviceInput, int *d_deviceOutput, int numElements)
{
	//TODO: Modify this to complete the functionality of the sort on the deivce
	int *swap_T, *T;
	dim3 block(BLOCK_SIZE, 1);
	dim3 grid((numElements / BLOCK_SIZE) + 1, 1);
	hipMalloc(&T, sizeof(int)*numElements);
	
	for (int bit = 0; bit < 15; bit++){
		split_A << <grid, block >> >(d_deviceInput, d_deviceOutput, numElements, bit);
		hipDeviceSynchronize();

		recursive_scan(T, d_deviceOutput, numElements);
		hipDeviceSynchronize();

		split_B << <grid, block >> >(T, d_deviceOutput, numElements);
		hipDeviceSynchronize();

		scatter << <grid, block >> >(d_deviceInput, T, d_deviceOutput, numElements);
		hipDeviceSynchronize();

		//swap
		swap_T = d_deviceInput;
		d_deviceInput = d_deviceOutput;
		d_deviceOutput = swap_T;

	}

}


int main(int argc, char **argv) {
	wbArg_t args;
	int *hostInput;  // The input 1D list
	int *hostOutput; // The output list
	int *deviceInput;
	int *deviceOutput;
	int numElements; // number of elements in the list

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput = (int *)wbImport(wbArg_getInputFile(args, 0), &numElements, "integral_vector");
	hipHostAlloc(&hostOutput, numElements * sizeof(int), hipHostMallocDefault);
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The number of input elements in the input is ", numElements);

	wbTime_start(GPU, "Allocating GPU memory.");
	wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(int)));
	wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(int)));
	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Clearing output memory.");
	wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(int)));
	wbTime_stop(GPU, "Clearing output memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(int),
		hipMemcpyHostToDevice));
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	wbTime_start(Compute, "Performing CUDA computation");
	sort(deviceInput, deviceOutput, numElements);
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
		hipMemcpyDeviceToHost));
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	hipFree(deviceInput);
	hipFree(deviceOutput);
	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, hostOutput, numElements);

	free(hostInput);
	hipHostFree(hostOutput);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}
