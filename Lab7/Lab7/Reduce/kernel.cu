#include "hip/hip_runtime.h"
// Sandip Saha Joy
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];

#include "hip/hip_runtime.h"
#include ""

#include <wb.h>

#define BLOCK_SIZE 512 // TODO: You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
			    }                                                                     \
        } while (0)

__global__ void total(float *input, float *output, int len) {
	// TODO: Load a segment of the input vector into shared memory
	// TODO: Traverse the reduction tree
	// TODO: Write the computed sum of the block to the output vector at the correct index

	__shared__ float partialSum[2 * BLOCK_SIZE];

	// each thread loads one element from global to shared mem
	unsigned int t = threadIdx.x;
	unsigned int start = 2 * blockIdx.x * BLOCK_SIZE;

	if (start + t < len){
		partialSum[t] = input[start + t];
	}
	else{
		partialSum[t] = 0.0;
	}
		
	if (start + t + BLOCK_SIZE < len){
		partialSum[t + BLOCK_SIZE] = input[start + t + BLOCK_SIZE];
	}
	else {
		partialSum[t + BLOCK_SIZE] = 0.0;
	}
		
	__syncthreads();


	// do reduction in shared mem
	for (int stride = BLOCK_SIZE; stride > 0; stride /= 2)
	{
		__syncthreads();
		if (t < stride)
		{
			partialSum[t] += partialSum[t + stride];
		}
	}


	// write result for this block to global mem
	if (t == 0)
	{
		output[blockIdx.x] = partialSum[0];
	}
	
}

int main(int argc, char **argv) {
	wbArg_t args;
	float *hostInput;  // The input 1D list
	float *hostOutput; // The output list
	float *deviceInput;
	float *deviceOutput;
	int numInputElements;  // number of elements in the input list
	int numOutputElements; // number of elements in the output list

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput =
		(float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);

	numOutputElements = numInputElements / (BLOCK_SIZE << 1);
	if (numInputElements % (BLOCK_SIZE << 1)) {
		numOutputElements++;
	}
	hostOutput = (float *)malloc(numOutputElements * sizeof(float));

	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The number of input elements in the input is ",
		numInputElements);
	wbLog(TRACE, "The number of output elements in the input is ",
		numOutputElements);

	wbTime_start(GPU, "Allocating GPU memory.");
	// TODO: Allocate GPU memory here
	hipMalloc(&deviceInput, sizeof(float)*numInputElements);
	hipMalloc(&deviceOutput, sizeof(float)*numOutputElements);
	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	// TODO: Copy memory to the GPU here
	hipMemcpy(deviceInput, hostInput, sizeof(float)*numInputElements, hipMemcpyHostToDevice);
	hipMemcpy(deviceOutput, hostOutput, sizeof(float)*numOutputElements, hipMemcpyHostToDevice);

	wbTime_stop(GPU, "Copying input memory to the GPU.");
	// TODO: Initialize the grid and block dimensions here
	dim3 dimGrid(((numInputElements + 1) / (BLOCK_SIZE) + 1), 1, 1);
	dim3 dimBlock(BLOCK_SIZE, 1);

	wbTime_start(Compute, "Performing CUDA computation");
	// TODO: Launch the GPU Kernel here
	total << <dimGrid, dimBlock >> >(deviceInput, deviceOutput, numInputElements);

	hipDeviceSynchronize();
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	// TODO: Copy the GPU memory back to the CPU here
	hipMemcpy(hostOutput, deviceOutput, sizeof(float)*numOutputElements, hipMemcpyDeviceToHost);
	wbTime_stop(Copy, "Copying output memory to the CPU");

	// TODO:
	/********************************************************************
	* Reduce output vector on the host
	* NOTE: One could also perform the reduction of the output vector
	* recursively and support any size input. For simplicity, we do not
	* require that for this lab.
	********************************************************************/
	int x = 1;
	while (x < numOutputElements)
	{
		hostOutput[0] += hostOutput[x];
		x++;
	}

	wbTime_start(GPU, "Freeing GPU Memory");
	// TODO: Free the GPU memory here
	hipFree(deviceInput);
	hipFree(deviceOutput);

	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, hostOutput, 1);

	free(hostInput);
	free(hostOutput);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}
