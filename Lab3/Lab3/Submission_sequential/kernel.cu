#include "hip/hip_runtime.h"
// Sandip Saha Joy

#include <wb.h>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__device__ int binarySearch(const int value, const int *A, const int N)
{
	// TODO: Implement a binary search that returns
	// the index where all values in A are less than
	// the given value.
	
	int first = 0, mid, last = N - 1;

	while (first <= last)
	{
		// Find mid
		mid = (first + last) / 2;

		// if value is greater, ignore first-mid half
		if (A[mid] < value)
		{
			first = mid + 1;
		}

		// if value is smaller, ignore mid-last half
		else
		{
			last = mid - 1;
		}
			
	}
	if (first < N)
		return first;
	else
		return N;

}

__device__ int linearSearch(const int value, const int *A, const int N)
{
	// TODO: Implement a sequential search that returns
	// the index where all values in A are less than
	// the given value.
	int i;

	for (i = 0; i < N; i++)
	{
		if (A[i] < value)
			continue;
		else
			break;
	}
	if (i < N)
		return i;
	else
		return N;
}

__global__ void merge(int *C, const int *A, const int *B, const int N)
{
	// TODO: Merge arrays A and B into C. To make it
	// easier you can assume the following:
	// 
	// 1) A and B are both size N
	//
	// 2) C is size 2N
	//
	// 3) Both A and B are sorted arrays
	//
	// The algorithm should work as follows:
	// Given inputs A and B as follows:
	// A = [0 2 4 10]
	// B = [1 5 7 9]
	//
	// Step 1:
	// Find for each element in array A the index i that
	// would A[i] be inserted in array B or in other 
	// words find the smallest j where A[i] < B[j].
	//
	// Step 2:
	// Do the same for B, but this time find the j 
	// where B[i] < A[j].
	//
	// Step 3:
	// Since we know how many elements come before
	// A[i] in array A and we know how many elements 
	// come before A[i] in array B, which is given by
	// are calculation of j. We should know where A[i]
	// is inserted into C, given i and j.
	//
	// This same logic can be used to find where B[i]
	// should be inserted into C. Although you will have
	// to make a minor change to handle duplicates in A 
	// and B. Or in other words if A and B intersect at 
	// all some values in C will be incorrect. This 
	// occurs because A and B will want to put the values 
	// in the same place in C.
	int x, y, i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N)
	{
		x = linearSearch(A[i], B, N);
		y = linearSearch(B[i]+1, A, N);
		C[i + x] = A[i];
		C[i + y] = B[i];
	}
	
}

int main(int argc, char **argv) {
	wbArg_t args;
	int N;
	int* A;
	int* B;
	int* C;
	int* deviceA;
	int* deviceB;
	int* deviceC;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	A = (int *)wbImport(wbArg_getInputFile(args, 0), &N, NULL, "Integer");
	B = (int *)wbImport(wbArg_getInputFile(args, 1), &N, NULL, "Integer");
	C = (int *)malloc(2 * N * sizeof(int));
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The input length is ", N);

	int threads = 256;
	int blocks = N / threads + ((N%threads == 0) ? 0 : 1);

	wbTime_start(GPU, "Allocating GPU memory.");
	hipMalloc((void **)&deviceA, N * sizeof(int));
	hipMalloc((void **)&deviceB, N * sizeof(int));
	hipMalloc((void **)&deviceC, 2 * N * sizeof(int));
	wbTime_stop(GPU, "Allocating GPU memory.");


	wbTime_start(GPU, "Copying input memory to the GPU.");
	hipMemcpy(deviceA, A, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceB, B, N * sizeof(int), hipMemcpyHostToDevice);
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	// Perform on CUDA.
	const dim3 blockSize(threads, 1, 1);
	const dim3 gridSize(blocks, 1, 1);

	wbTime_start(Compute, "Performing CUDA computation");
	merge << < gridSize, blockSize >> >(deviceC, deviceA, deviceB, N);
	hipDeviceSynchronize();
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	hipMemcpy(C, deviceC, 2 * N * sizeof(int), hipMemcpyDeviceToHost);
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);
	wbTime_stop(GPU, "Freeing GPU Memory");

	for (int i = 0; i < 2 * N; i++){
		printf("%d ", C[i]);
	}

	wbSolution(args, C, 2*N);

	free(A);
	free(B);
	free(C);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}
















































































// Reference: https://bit.ly/31nxbmi