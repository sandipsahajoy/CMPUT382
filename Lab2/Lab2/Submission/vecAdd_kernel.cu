#include "hip/hip_runtime.h"
// Sandip Saha Joy
// Kernel for adding to vectors
// There are a few TODOs that you need to fill out
#include "hip/hip_runtime.h"
#include ""

#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
	// TODO: Insert code to implement vector addition here
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if (i<len) out[i] = in1[i] + in2[i];
}

int main(int argc, char **argv) {
	wbArg_t args;
	int inputLength;
	float *hostInput1;
	float *hostInput2;
	float *hostOutput;
	float *deviceInput1;
	float *deviceInput2;
	float *deviceOutput;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput1 = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
	hostInput2 = (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
	hostOutput = (float *)malloc(inputLength * sizeof(float));
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The input length is ", inputLength);
	int size = inputLength*sizeof(float);

	wbTime_start(GPU, "Allocating GPU memory.");
	// TODO: Allocate GPU memory here
	hipMalloc((void**)&deviceInput1, size);
	hipMalloc((void**)&deviceInput2, size);
	hipMalloc((void**)&deviceOutput, size);
	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	// TODO: Copy memory to the GPU here
	hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
	hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	// TODO: Initialize the grid and block dimensions here
	// HINT: Look up CUDAs dim3 
	dim3 DimGrid((inputLength - 1) / 256 + 1, 1, 1);
	dim3 DimBlock(256, 1, 1);
	// Optional you might want to log the block and grid
	// dimensions as follows:
	// wbLog(TRACE, "Block dimension is ", blockDim.x);
	// wbLog(TRACE, "Grid dimension is ", gridDim.x);
	// Don't uncomment the above code, they do not work on the machines in the lab

	wbTime_start(Compute, "Performing CUDA computation");
	// TODO: Launch the GPU Kernel here
	vecAdd<<<DimGrid, DimBlock>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	// TODO: Copy the GPU memory back to the CPU here
	hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	// TODO: Free the GPU memory here
	hipFree(deviceInput1);
	hipFree(deviceInput2);
	hipFree(deviceOutput);
	wbTime_stop(GPU, "Freeing GPU Memory");

	hipDeviceSynchronize();

	wbSolution(args, hostOutput, inputLength);

	free(hostInput1);
	free(hostInput2);
	free(hostOutput);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}

